#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#define N 250
#define threads 32

__global__ void kernel(int *mat, int *vec)
{
    //__shared__ int sh[threads*threads];
    int value;
    __shared__ int Qsh[threads], Psh[threads];
    int P = 0, Q = 0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int indexMat = col + row * N;
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;

    if (col >= N || row >= N) return;

    Qsh[threadIdx.y] = 0;
    Psh[threadIdx.y] = 0;

    value = mat[indexMat];
    if (value > 0) {
        P = 1;
    } else {
        Q = 1;
    }

    atomicAdd(&Psh[threadIdx.y], P);
    atomicAdd(&Qsh[threadIdx.y], Q);

    __syncthreads();

    if (threadIdx.x == 0)
        atomicAdd(&vec[row], Psh[threadIdx.y]);
   
    __syncthreads();
   
    if (threadIdx.x == 0)
        atomicSub(&vec[row], Qsh[threadIdx.y]);

    __syncthreads();

    if (value < 0) {
        value = vec[row];
        mat[indexMat] = value;
    }

}

void PrintMat(int* mat,int n,int m) {
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
            printf("%d ", mat[i * N + j]);

        printf("\n");
    }
}

void hostkernel(int* A,int* B)
{
    for(int i = 0; i < N; i++) {
        int pos = 0, neg = 0;
        for (int j = 0; j < N; j++)
            if (A[i * N + j] > 0) pos++; else neg++;
        
        B[i] = pos - neg;

        for (int j = 0; j < N; j++)
            if (A[i * N + j] < 0) A[i * N + j] = B[i];
    }
}

void compare(int* A, int* B)
{
    for (int i = 0; i < N; i++)
        if (A[i] != B[i])
            printf("nisu isti");
}

int main()
{
    int A[N * N], B[N], Bh[N],A2[N*N];
    int* Ad, * Bd;

    hipMalloc((void**)&Ad, sizeof(int) * N * N);
    hipMalloc((void**)&Bd, sizeof(int) * N);
    
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            int val = rand() % 10;
            A[i * N + j] = val % 3 == 0 ? -val : val;
        }
    for (int i = 0; i < N; i++)
        B[i] = 0;

    printf("===== A =====\n");
    //PrintMat(A, N, N);
    printf("=====  =====\n");

    hipMemcpy(Ad, A, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, sizeof(int) * N, hipMemcpyHostToDevice);
    
    dim3 gridSize((N + threads - 1) / threads, (N + threads - 1) / threads);
    dim3 blockSize(threads, threads);

    kernel << <gridSize, blockSize >> > (Ad, Bd);

    hipMemcpy(A2, Ad, sizeof(int) * N * N, hipMemcpyDeviceToHost);
    hipMemcpy(B, Bd, sizeof(int) * N, hipMemcpyDeviceToHost);
    hostkernel(A, Bh);


   /* printf("===== DEVICE =====\n");
    PrintMat(B,1,N);
    printf("=====  =====\n");
    PrintMat(A2, N, N);
    printf("===== HOST =====\n");
    PrintMat(Bh, 1, N);
    printf("=====  =====\n");
    PrintMat(A, N, N);*/

    compare(B, Bh);
    
    return 0;
}