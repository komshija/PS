#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N 50
#define threads 6 
// radi za 6, za 7 threads ne radi


__global__ void kernel(int* A, int* B, int n)
{
	extern __shared__ int sh[]; // velicina (threads + 2)^2
	int res;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int indA = col + row * n;
	int indB = col + row * (n - 2);

	int red = threads + 2;
	int threadInd = threadIdx.x + threadIdx.y * red;

	if (col >= n && row >= n) return;

	sh[threadInd] = A[indA];
	//desno dva
	if (threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1)
	{
		sh[threadInd + 1] = A[indA + 1];
		sh[threadInd + 2] = A[indA + 2];
	}
	if (threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1)
	{
		sh[threadInd + red] = A[indA + n];
		sh[threadInd + red * 2] = A[indA + 2 * n];
	}
	if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1
		&& blockIdx.x != gridDim.x - 1 && blockIdx.y != gridDim.y - 1)
	{
		sh[threadInd + red + 1] = A[indA + n + 1];
		sh[threadInd + red + 2] = A[indA + n + 2];
		sh[threadInd + 1 + red * 2] = A[indA + 1 + 2 * n];
		sh[threadInd + 2 + red * 2] = A[indA + 2 + 2 * n];
	}

	__syncthreads();

	if (row < n - 2 && col < n - 2) {

		res = sh[threadInd] +
			sh[threadInd + 1] + sh[threadInd + 2] +
			sh[threadInd + red] + sh[threadInd + red * 2]
			+ sh[threadInd + red + 1] + sh[threadInd + red + 2]
			+ sh[threadInd + 1 + 2 * red] + sh[threadInd + 2 + 2 * red];
		res /= 9.f;

		B[indB] = res;
	}



}

void hostkernel(int* A, int* B)
{
	for (int i = 0; i < N - 2; i++) {
		for (int j = 0; j < N - 2; j++) {
			B[i * (N - 2) + j] = 
				  A[(i)*N + (j)]  
				+ A[(i + 1) * N + (j)] + A[(i + 2) * N + (j)]
				+ A[(i)*N + (j + 1)] + A[(i)*N + (j + 2)]
				+ A[(i + 1) * N + (j + 1)] + A[(i + 1) * N + (j + 2)]
				+ A[(i + 2) * N + (j + 1)] + A[(i + 2) * N + (j + 2)];
			B[i * (N - 2) + j] /= 9.f;

		}
	}
}

void PrintMat(int* B) {
	for (int i = 0; i < N - 2; i++) {
		for (int j = 0; j < N - 2; j++) {
			printf("%d ", B[i * (N - 2) + j]);
		}
		printf("\n");
	}
}

void printmat(int* A, int n, int m) {
	for (int i = 0; i < n; i++) {
		for (int k = 0; k < m; k++) {
			printf("%d ", A[i * m + k]);
		}
		printf("\n");
	}
}

int main()
{
	int A[N * N], B[(N - 2) * (N - 2)], Bh[(N - 2) * (N - 2)];
	int* Ad, * Bd;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			A[i * N + j] = 10;
		}
	}
	printmat(A, N , N);

	hipMalloc((void**)&Ad, sizeof(int) * N * N);
	hipMalloc((void**)&Bd, sizeof(int) * (N - 2) * (N - 2));

	hipMemcpy(Ad, A, sizeof(int) * N * N, hipMemcpyHostToDevice);

	dim3 gridSize((N + threads - 1) / threads, (N + threads - 1) / threads);
	dim3 blockSize(threads, threads);
	int memSize = (threads + 2) * (threads + 2);

	kernel << <gridSize, blockSize, memSize >> > (Ad, Bd, N);
	hipMemcpy(B, Bd, sizeof(int) * (N - 2) * (N - 2), hipMemcpyDeviceToHost);

	hostkernel(A, Bh);

	printf("\nHost\n");
	PrintMat(Bh);
	printf("\nDevice\n");
	PrintMat(B);

	hipFree(Ad);
	hipFree(Bd);

	return 0;
}