#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#define N 50
#define M 50
#define threads 32

__global__ void kernel(int* mat, int* k, int* kmat)
{
	__shared__ int sh[(threads + 1) * (threads + 1)];
	__shared__ int ksh[9];
	__shared__ int ksum;
	ksum = 0;
	int res;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int index = col + row * M;
	int threadIndex = threadIdx.x + threadIdx.y * (blockDim.x + 1);

	if (blockIdx.x > 0)
		threadIndex++;
	if (blockIdx.y > 0)
		threadIndex += blockDim.x + 1;

	if (threadIdx.x < 3 && threadIdx.y < 3)
	{
		ksh[threadIdx.x + threadIdx.y * 3] = k[threadIdx.x + threadIdx.y * 3];
		atomicAdd(&ksum, ksh[threadIdx.x + threadIdx.y * 3]);
	}

	if (row >= N || col >= M) return;

	sh[threadIndex] = mat[index];

	//desno
	if (threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1)
		sh[threadIndex + 1] = mat[index + 1];
	//dole
	if (threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1)
		sh[threadIndex + blockDim.x + 1] = mat[index + M];
	//gore
	if (blockIdx.y > 0 && threadIdx.y == 0)
		sh[threadIndex - blockDim.x - 1] = mat[index - M];
	//levo
	if (blockIdx.x > 0 && threadIdx.x == 0)
		sh[threadIndex - 1] = mat[index - 1];

	//gore levo
	if (blockIdx.y > 0 && threadIdx.y == 0 && blockIdx.x != gridDim.x - 1 && blockIdx.x > 0 && threadIdx.x == 0)
		sh[threadIndex - 1 - blockDim.x - 1] = mat[index - 1 - M];
	//dole desno
	if (threadIdx.x == blockDim.x - 1 && blockIdx.y != gridDim.y - 1 && threadIdx.y == blockDim.y - 1)
		sh[threadIndex + blockDim.x + 2] = mat[index + M + 1];
	//gore desno
	if (threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1 && blockIdx.y > 0 && threadIdx.y == 0)
		sh[threadIndex - blockDim.x] = mat[index + 1 - M];
	//dole levo
	if (threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1 && blockIdx.x > 0 && threadIdx.x == 0)
		sh[threadIndex + blockDim.x] = mat[index + M - 1];

	__syncthreads();

	if (index % M == 0 || index < M || (index + 1) % M == 0 || index >= (N - 1) * M)
		res = sh[threadIndex];
	else {
		res = 0;
		int startIndex = threadIndex - 1 - (blockDim.x + 1);
		for (int i = 0; i < 3; i++) {
			for (int k = 0; k < 3; k++) {
				res += sh[startIndex + k] * ksh[i * 3 + k];
			}
			startIndex += blockDim.x + 1;
		}
		res /= (float)ksum;
	}

	kmat[index] = res;
}

void hostkernel(int* A, int* K, int* AR)
{
	int ksum = 0;
	for (int i = 0; i < 9; i++)
		ksum += K[i];

	for (int i = 0; i < N; i++) {
		for (int k = 0; k < M; k++) {
			if (i == 0 || k == 0 || i == N - 1 || k == M - 1)
				AR[i * M + k] = A[i * M + k];
			else {
				AR[i * M + k] = 0;
				for (int p = -1; p < 2; p++)
					for (int q = -1; q < 2; q++)
						AR[i * M + k] += A[(i + p) * M + (k + q)] * K[(1 + p) * 3 + (1 + q)];
				AR[i * M + k] /= ksum;
			}
		}
	}
}

void printmat(int* A, int n, int m) {
	for (int i = 0; i < n; i++) {
		for (int k = 0; k < m; k++) {
			printf("%d ", A[i * m + k]);
		}
		printf("\n");
	}
}

bool compare(int* A, int* B, int max)
{
	for (int i = 0; i < max; i++)
	{
		if (A[i] != B[i])
			return false;
	}
	return true;
}

int main()
{
	//Za identicno resenje kao na blanketu pozvati kernel za tri matrice tj R, G i B
	int A[N * M], K[9], AR[N * M], ARh[N * M];
	int* Ad, * Kd, * ARd;

	hipMalloc((void**)&Ad, sizeof(int) * N * M);
	hipMalloc((void**)&ARd, sizeof(int) * N * M);
	hipMalloc((void**)&Kd, sizeof(int) * 9);

	for (int i = 0; i < N; i++)
		for (int j = 0; j < M; j++)
			A[i * M + j] = rand() % 15;
	for (int i = 0; i < 9; i++)
		K[i] = rand() % 10;

	printf("===== A =====\n");
	printmat(A, N, M);
	/*
	printf("===== K =====\n");
	printmat(K,3,3);
	printf("===== = =====\n\n");*/

	hipMemcpy(Ad, A, sizeof(int) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(Kd, K, sizeof(int) * 9, hipMemcpyHostToDevice);

	dim3 gridSize((N + threads - 1) / threads, (M + threads - 1) / threads);
	dim3 blockSize(threads, threads);

	kernel << <gridSize, blockSize >> > (Ad, Kd, ARd);

	hipMemcpy(AR, ARd, sizeof(int) * N * M, hipMemcpyDeviceToHost);

	hostkernel(A, K, ARh);

	printf("===== AR =====\n");
	printmat(AR, N, M);
	printf("===== ARh =====\n");
	printmat(ARh, N, M);
	printf("===== = =====\n");

	bool res = compare(AR, ARh, N * M);
	if (res)
		printf("Isti");

	hipFree(Ad);
	hipFree(Kd);
	hipFree(ARd);

	return 0;
}
