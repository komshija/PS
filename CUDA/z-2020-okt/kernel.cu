﻿
#include <stdio.h>
#include "hip/hip_runtime.h"

#define N 9
#define threads 3

__global__ void mulKernel(int* A, int* X, int* B)
{
    __shared__ int a[threads * threads];
    __shared__ int x[threads];
    __shared__ int parcial[threads];
    int res = 0;

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int iA = col + row * N;

    int tidA = threadIdx.x + threads * threadIdx.y;
    int tidX = threadIdx.y;

    if (row >= N || col >= N) return;

    a[tidA] = A[iA];
    if (threadIdx.x == 0)
        x[tidX] = B[row];

    __syncthreads();

    res = a[tidA] * x[tidX];
    atomicAdd(&parcial[tidX], res);
    
    __syncthreads();

    if (threadIdx.x == 0)
        atomicAdd(&B[col], parcial[tidX]);

}

//lab2-6
__global__ void kernel6(int* A, int* posmin)
{
    //__shared__ int sh[threads];
    __shared__ int locPosMin;
    int loc;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    loc = A[index];

    if (threadIdx.x == 0)
        locPosMin = INT_MAX;
    __syncthreads();
    
    if(loc > 0)
        atomicMin(&locPosMin, loc);
    __syncthreads();
    
    if (threadIdx.x == 0)
        atomicMin(posmin, locPosMin);
    __syncthreads();
    
    if (threadIdx.x == 0)
        locPosMin = *posmin;
    __syncthreads();

    if (loc <= 0)
        loc = locPosMin;
    A[index] = loc;
}

int main()
{
    

    return 0;
}