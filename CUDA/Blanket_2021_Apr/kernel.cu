#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>

#define N 7
#define threads 2

__global__ void avgKernel(int* A, int* B)
{
	__shared__ int sh[(threads + 2) * (threads + 2)];
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int indexA = row * N + col;
	int indexB = row * (N - 2) + col;
	int red = threads + 2;
	int indexThread = threadIdx.x + threadIdx.y * red;
	int sum = 0;
	if (row >= N || col >= N) return;


	sh[indexThread] = A[indexA];
	if (threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1)
	{
		sh[indexThread + 1] = A[indexA + 1];
		sh[indexThread + 2] = A[indexA + 2];
	}
	if (threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1)
	{
		sh[indexThread + red] = A[indexA + N];
		sh[indexThread + 2 * red] = A[indexA + 2 * N];
	}
	if (threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1
		&& threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1)
	{
		sh[indexThread + 1 + red] = A[indexA + 1 + N];
		sh[indexThread + 2 + red] = A[indexA + 2 + N];
		sh[indexThread + 1 + 2 * red] = A[indexA + 1 + 2 * N];
		sh[indexThread + 2 + 2 * red] = A[indexA + 2 + 2 * N];
	}

	__syncthreads();

	if (col >= N - 2 || row >= N - 2) return;

	sum = sh[indexThread] + sh[indexThread + 2] + sh[indexThread + 2 * red] + sh[indexThread + 2 * red + 2];
	sum /= 4.f;
	B[indexB] = sum;

}

void avgHost(int* A, int* B)
{
	for (int i = 0; i < N - 2; i++)
		for (int j = 0; j < N - 2; j++)
			B[i * (N - 2) + j] = (A[i * N + j] + A[i * N + j + 2] + A[i * N + j + 2*N] + A[i * N + 2 + j + 2 * N]) / 4.f;
}

void Print(int* mat, int n, int m)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < m; j++)
		{
			printf("%d ", mat[i * m + j]);
		}
		printf("\n");
	}
}

int main()
{
	int A[N * N], B[(N - 2) * (N - 2)], Bh[(N - 2) * (N - 2)];
	int* Ad, * Bd;

	srand(0);
	for (int i = 0; i < N * N; i++)
		A[i] = i;
	printf("A\n");
	Print(A, N, N );

	hipMalloc((void**)&Ad, sizeof(int) * N * N);
	hipMalloc((void**)&Bd, sizeof(int) * (N - 2) * (N - 2));
	hipMemcpy(Ad, A, sizeof(int) * N * N, hipMemcpyHostToDevice);

	dim3 gridSize((N - 1 + threads) / threads, (N - 1 + threads) / threads);
	dim3 blockSize(threads, threads);

	avgKernel << <gridSize, blockSize >> > (Ad, Bd);

	hipMemcpy(B, Bd, sizeof(int) * (N - 2) * (N - 2), hipMemcpyDeviceToHost);
	avgHost(A, Bh);

	printf("HOST\n");
	Print(Bh, N - 2, N - 2);
	printf("DEVICE\n");
	Print(B, N - 2, N - 2);

	hipFree(Ad);
	hipFree(Bd);

	return 0;
}