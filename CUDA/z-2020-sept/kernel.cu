#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N 6
#define threads 2

__global__ void kernel(int* A, int* B)
{
	const int red = threads + 2;
	__shared__ int sh[red * red];
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int index = col + row * N;
	int indexB = col + row * (N - 2);
	int threadIndex = threadIdx.x + threadIdx.y * red;
	float res = 0;

	if (col >= N || row >= N) return;

	sh[threadIndex] = A[index];
	//dva desno
	if (threadIdx.x == blockDim.x - 1 && blockIdx.x != gridDim.x - 1)
	{
		sh[threadIndex + 1] = A[index + 1];
		sh[threadIndex + 2] = A[index + 2];
	}
	if (threadIdx.y == blockDim.y - 1 && blockIdx.y != gridDim.y - 1)
	{
		sh[threadIndex + red] = A[index + N];
		sh[threadIndex + 2 * red] = A[index + 2 * N];
	}
	if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1)
	{
		sh[threadIndex + red + 1] = A[index + N + 1];
		sh[threadIndex + 2 * red + 1] = A[index + 2 * N + 1];
		sh[threadIndex + red + 2] = A[index + N + 2];
		sh[threadIndex + 2 * red + 2] = A[index + 2 * N + 2];
	}

	__syncthreads();
	if (col >= N - 2 || row >= N - 2) return;

	res = sh[threadIndex]
		+ sh[threadIndex + 1] + sh[threadIndex + 2]
		+ sh[threadIndex + red] + sh[threadIndex + 2 * red]
		+ sh[threadIndex + red + 1] + sh[threadIndex + red + 2]
		+ sh[threadIndex + 2 * red + 1] + sh[threadIndex + 2 * red + 2];
	res /= 9.f;
	B[indexB] = res;

}

void hostkernel(int* A, int* B)
{
	for (int i = 0; i < N - 2; i++) {
		for (int j = 0; j < N - 2; j++) {
			B[i * (N - 2) + j] =
				A[(i)*N + (j)]
				+ A[(i + 1) * N + (j)] + A[(i + 2) * N + (j)]
				+ A[(i)*N + (j + 1)] + A[(i)*N + (j + 2)]
				+ A[(i + 1) * N + (j + 1)] + A[(i + 1) * N + (j + 2)]
				+ A[(i + 2) * N + (j + 1)] + A[(i + 2) * N + (j + 2)];
			B[i * (N - 2) + j] /= 9.f;

		}
	}
}

void PrintMat(int* B) {
	for (int i = 0; i < N - 2; i++) {
		for (int j = 0; j < N - 2; j++) {
			printf("%d ", B[i * (N - 2) + j]);
		}
		printf("\n");
	}
}

void printmat(int* A, int n, int m) {
	for (int i = 0; i < n; i++) {
		for (int k = 0; k < m; k++) {
			printf("%d ", A[i * m + k]);
		}
		printf("\n");
	}
}

int main()
{
	int A[N * N], B[(N - 2) * (N - 2)], Bh[(N - 2) * (N - 2)];
	int* Ad, * Bd;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			A[i * N + j] = 10;
		}
	}
	printmat(A, N, N);

	hipMalloc((void**)&Ad, sizeof(int) * N * N);
	hipMalloc((void**)&Bd, sizeof(int) * (N - 2) * (N - 2));

	hipMemcpy(Ad, A, sizeof(int) * N * N, hipMemcpyHostToDevice);

	dim3 gridSize((N + threads - 1) / threads, (N + threads - 1) / threads);
	dim3 blockSize(threads, threads);
	int memSize = (threads + 2) * (threads + 2);

	kernel << <gridSize, blockSize, memSize >> > (Ad, Bd, N);
	hipMemcpy(B, Bd, sizeof(int) * (N - 2) * (N - 2), hipMemcpyDeviceToHost);

	hostkernel(A, Bh);

	printf("\nHost\n");
	PrintMat(Bh);
	printf("\nDevice\n");
	PrintMat(B);

	hipFree(Ad);
	hipFree(Bd);

	return 0;
}