#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N 100
#define THREADS 32

__global__ void scalaradd(int* A, int* B, int* C)
{
    __shared__ int a[THREADS];
    __shared__ int b[THREADS];
    __shared__ int sum;
    sum = 0;
    int proizvod;
    int ind = threadIdx.x + blockDim.x * blockIdx.x;

    if (ind < N) {
        a[threadIdx.x] = A[ind];
        b[threadIdx.x] = B[ind];
    }

    if (ind < N)
        proizvod = a[threadIdx.x] * b[threadIdx.x];
    else
        proizvod = 0;

    atomicAdd(&sum, proizvod);
    __syncthreads();

    if (threadIdx.x == 0)
        atomicAdd(C, sum);

}


int skalar(int* A, int* B) {
    int sum = 0;
    for (int i = 0; i < N; i++) {
        sum += A[i] * B[i];
    }
    return sum;
}

int main()
{
    int A[N], B[N], C, Ch;
    int* Ad, * Bd, * Cd;
    int gridDim = (N + THREADS - 1) / THREADS;
    C = 0;
    hipMalloc((void**)&Ad, sizeof(int) * N);
    hipMalloc((void**)&Bd, sizeof(int) * N);
    hipMalloc((void**)&Cd, sizeof(int));

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N / 5; j++) {
            A[i] = (j * i);
            B[i] = (j * i);
        }
        A[i] /= i % 4 == 0 ? 2 : 5;
        B[i] /= 2;
        B[i] *= i % 4 == 0 ? -1 : 2;
    }


    hipMemcpy(Ad, A, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(Cd, &C, sizeof(int), hipMemcpyHostToDevice);

    dim3 gridSize(gridDim);
    dim3 blockSize(THREADS);
    int memSize = 100;
    scalaradd << <gridSize, blockSize ,memSize >> > (Ad, Bd, Cd);

    hipMemcpy(&C, Cd, sizeof(int), hipMemcpyDeviceToHost);
    Ch = skalar(A, B);

    if (C == Ch) {

        printf("Isti rezultat. device = %d host = %d\n",C, Ch);
    }
    else {
        printf("Nije isti rezultat. device = %d host = %d\n",C, Ch);
    }
    
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);

    return 0;
}