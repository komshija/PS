#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N 40

__global__ void kernel(int* A, int* X, int* B, int num)
{
	__shared__ int sum[32];
	int proizvod;
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int indA = i * num + k;
	sum[threadIdx.y] = 0;

	__syncthreads();
	if (i < num && k < num)
		proizvod = A[indA] * X[k];
	else
		proizvod = 0;

	atomicAdd(&sum[threadIdx.y], proizvod);

	__syncthreads();

	if (threadIdx.x == 0)
		atomicAdd(&B[i],sum[threadIdx.y]);

}

void Hostkenrel(int* A, int* X, int* B)
{
	for (int i = 0; i < N; i++)
	{
		B[i] = 0;
		for (int k = 0; k < N; k++)
		{
			B[i] += A[i * N + k] * X[k];
		}
	}
}

void PrintVec(int* vec) {
	for (int i = 0; i < N; i++)
		printf("%d ", vec[i]);
	printf("\n");
}

int main()
{
	int A[N * N], X[N], B[N], Bh[N];
	int* Ad, * Xd, * Bd;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[i * N + j] = j + j;
			printf("%d ", A[i * N + j]);
		}
		X[i] = i;
		B[i] = 0;
		printf("\n");
	}
	printf("\n");

	PrintVec(X);

	hipMalloc((void**)&Ad, sizeof(int) * N * N);
	hipMalloc((void**)&Xd, sizeof(int) * N);
	hipMalloc((void**)&Bd, sizeof(int) * N);

	hipMemcpy(Ad, A, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(Xd, X, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, sizeof(int) * N, hipMemcpyHostToDevice);

	dim3 gridSize((N + 31) / 32, (N + 31) / 32);
	dim3 blockSize(32, 32);

	kernel << <gridSize, blockSize >> > (Ad, Xd, Bd, N);

	hipMemcpy(B, Bd, sizeof(int) * N, hipMemcpyDeviceToHost);
	Hostkenrel(A, X, Bh);

	PrintVec(B);
	printf("\n");
	PrintVec(Bh);

	hipFree(Ad);
	hipFree(Bd);
	hipFree(Xd);

	return 0;
}