#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define THREADS 32

__global__ void func(int* A, int* B, int N)
{
	extern __shared__ int sh[];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) return;

	int* Ash = sh;
	int* res = &sh[N + 2];

	Ash[threadIdx.x] = A[tid];
	// Fora je u tome da ako je nit poslednja u bloku mora uzme jos 2 preko
	// Ali ako je globalni indeks niti poslednji u nizu, takodje mora da uzme jos 2 preko
	if (threadIdx.x == blockDim.x - 1 || tid == N - 1) {
		Ash[threadIdx.x + 1] = A[tid + 1];
		Ash[threadIdx.x + 2] = A[tid + 2];
	}
	__syncthreads();

	res[threadIdx.x] = (3 * Ash[threadIdx.x] + 10 * Ash[threadIdx.x + 1] + 7 * Ash[threadIdx.x + 2]) / 20.f;
	__syncthreads();

	B[tid] = res[threadIdx.x];

}
void Host_func(int* A, int* B, int N) {
	for (int i = 0; i < N; i++) {
		B[i] = (3 * A[i] + 10 * A[i + 1] + 7 * A[i + 2]) / 20.f;
	}
}

int CompareArrays(int* A, int* B,int N) {
	int res = 0;
	for (int i = 0; i < N; i++) {
		if (A[i] != B[i]) res++;
	}
	return res;
}

void Print(int* A, int N) {
	for (int i = 0; i < N; i++)
		printf("%d ", A[i]);
	printf("\n");
}

int main()
{
	int* A, * B, N, * Ad, * Bd, * Bh;
	scanf("%d", &N);
	A = new int[N + 2];
	B = new int[N];
	Bh = new int[N];
	hipMalloc((void**)&Ad, sizeof(int) * (N+2));
	hipMalloc((void**)&Bd, sizeof(int) * N);

	for (int i = 0; i < N+2; i++) {
		A[i] = i;
	}
	hipMemcpy(Ad, A, sizeof(int) * (N+2), hipMemcpyHostToDevice);

	dim3 gridSize((N + (THREADS - 1)) / THREADS);
	dim3 blockSize(THREADS);

	func << <gridSize, blockSize, (N + 2 + N) * sizeof(int) >> > (Ad, Bd, N);

	Host_func(A, Bh, N);
	hipMemcpy(B, Bd, sizeof(int) * N, hipMemcpyDeviceToHost);
	int diff = CompareArrays(B, Bh, N);
	printf("difference %d\n", diff);
	Print(B, N);
	Print(Bh, N);

	hipFree(Ad);
	hipFree(Bd);

	return 0;
}
