#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#define N 14
#define M 7
#define threads 4

__global__ void sumMin(int* A, int* pom, int* min)
{
	__shared__ int locmin;
	__shared__ int sh[threads + M - 1];
	int sum = 0;
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index >= N) return;
	sh[threadIdx.x] = A[index];

	if (index >= N - M + 1) return;

	if (blockDim.x - 1 == threadIdx.x)
		for (int i = 1; i < M; i++)
			sh[threadIdx.x + i] = A[index + i];
	
	__syncthreads();
	for (int i = 0; i < M; i++)
		sum += sh[threadIdx.x + i];
	pom[index] = sum;

	if (threadIdx.x == 0 && blockIdx.x == 0)
		*min = sum;
	
	if (threadIdx.x == 0)
		locmin = sum;
	__syncthreads();

	atomicMin(&locmin, sum);
	__syncthreads();

	if (threadIdx.x == 0)
		atomicMin(min, locmin);

}

int main()
{
	int A[N], pom[N - M + 1], min;
	int* Ad, * pomd, * mind;

	printf("A\n");
	for (int i = 0; i < N; i++)
	{
		A[i] = rand() % 10;
		printf("%d ", A[i]);
	}
	printf("\n");

	hipMalloc((void**)&Ad, sizeof(int) * N);
	hipMalloc((void**)&pomd, sizeof(int) * (N - M + 1));
	hipMalloc((void**)&mind, sizeof(int));

	hipMemcpy(Ad, A, sizeof(int) * N, hipMemcpyHostToDevice);

	sumMin << <(N - M + threads) / threads, threads >> > (Ad, pomd, mind);

	hipMemcpy(pom, pomd, sizeof(int) * (N - M + 1), hipMemcpyDeviceToHost);
	hipMemcpy(&min, mind, sizeof(int), hipMemcpyDeviceToHost);

	printf("pom\n");
	for (int i = 0; i < N - M + 1; i++)
		printf("%d ", pom[i]);
	printf("\n");
	printf("Min= %d ", min);

	return 0;
}