#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 50
#define THREADS 32

__global__ void kernel(int* A, int* B)
{
	__shared__ int sh[(THREADS + 2) * (THREADS + 2)];
	int res,minimum,maximum;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int indexA = row * (N + 2) + col;
	int indexB = (row-1) * N + (col-1);

	int threadIndex = threadIdx.x + threadIdx.y * (THREADS + 2) + (THREADS + 2) + 1;
	if (row >= N + 2 || col >= N + 2) return;

	// ucitavanje podataka
	sh[threadIndex] = A[indexA];

	// desno
	if (threadIdx.x == blockDim.x - 1 && col != (N+1) ) 
		sh[threadIndex + 1] = A[indexA + 1];

	// dole
	if (threadIdx.y == blockDim.y - 1 && row != (N+1)) 
		sh[threadIndex + (THREADS + 2)] = A[indexA + N + 2]; 

	// levo
	if (col != 0 && threadIdx.x == 0 && blockIdx.x > 0)  
		sh[threadIndex - 1] = A[indexA - 1]; 

	// gore
	if (blockIdx.y > 0 && threadIdx.y == 0 && indexA % (N+2) != 0 )
		sh[threadIndex - (THREADS + 2)] = A[indexA - N - 2];
	

	__syncthreads();
	if (col < 1 || row < 1 || col > N || row > N) return;
	//izracunavanje

	minimum = min(sh[threadIndex - 1], sh[threadIndex + 1]);
	maximum = max(sh[threadIndex + (THREADS+2)], sh[threadIndex - (THREADS+2)]);
	res = maximum - minimum;

	B[indexB] = res;

}


void HostKernel(int* A, int* B)
{
	int k = 0;
	for (int i = 1; i < N + 1 ; i++) {
		for (int j = 1; j < N + 1; j++) {
			int max1 = A[(i - 1) * (N + 2) + j];
			int max2 = A[(i + 1) * (N + 2) + j];
			int min1 = A[i * (N + 2) + j - 1];
			int min2 = A[i * (N + 2) + j + 1];

			B[k] = max(max1, max2) - min(min1,min2);
			k++;
		}
	}
}

bool compare(int* A, int* B) {
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			if (A[i * N + j] != B[i * N + j]) return false;
	return true;
}

int main()
{
	int A[(N + 2) * (N + 2)], B[N * N], Bh[N * N];
	int* Ad, * Bd;

	hipMalloc((void**)&Ad, sizeof(int) * (N + 2) * (N + 2));
	hipMalloc((void**)&Bd, sizeof(int) * N * N);

	for (int i = 0; i < N + 2; i++) {
		for (int j = 0; j < N + 2; j++) {
			A[i * (N+2) + j] = rand() % 10;
			printf("%d ", A[i * (N+2) + j]);
		}
		printf("\n");
	}

	hipMemcpy(Ad, A, sizeof(int) * (N + 2) * (N + 2), hipMemcpyHostToDevice);

	dim3 gridSize(((N+2) + THREADS - 1) / THREADS, ((N + 2) + THREADS - 1) / THREADS);
	dim3 blockSize(THREADS, THREADS);

	kernel << <gridSize, blockSize >> > (Ad, Bd);

	hipMemcpy(B, Bd, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	HostKernel(A, Bh);

	printf("HOST\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
			printf("%d ", Bh[i * N + j]);
		printf("\n");
	}

	printf("DEVICE\n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
			printf("%d ", B[i * N + j]);
		printf("\n");
	}


	if (compare(B, Bh))
		printf("isti\n");
	else
		printf("razliciti\n");


	hipFree(Ad);
	hipFree(Bd);

	return 0;
}
