#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N 32
#define THREADS 4

__global__ void Expression(int* A, int x, int* B, int* C)
{
	extern __shared__ int niz[];
	__shared__ int sx;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid >= N) return;

	sx = x;
	int* sa = niz;
	int* sb = &niz[N];
	int* sc = &niz[N];

	sa[tid] = A[tid];
	sb[tid] = B[tid];

	sc[tid] = (sa[tid] * sx) + sb[tid];

	C[tid] = sc[tid];
}


void Host_Expression(int* A, int x, int* B, int* C) {
	for (int i = 0; i < N; i++) {
		C[i] = A[i] * x + B[i];
	}
}

int CompareArrays(int* A, int* B) {
	int res = 0;
	for (int i = 0; i < N; i++) {
		if (A[i] != B[i]) res++;
	}
	return res;
}

int main()
{
	int A[N], B[N], C[N], Ch[N], x;
	int* dA, * dB, * dC;
	x = 3;
	for (int i = 0; i < N; i++)
	{
		A[i] = i * i;
		B[i] = i + 2;
	}
	hipMalloc((void**)&dA, sizeof(int) * N);
	hipMalloc((void**)&dB, sizeof(int) * N);
	hipMalloc((void**)&dC, sizeof(int) * N);
	hipMemcpy(dA, A, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(int) * N, hipMemcpyHostToDevice);

	dim3 gridSize((N + (THREADS - 1)) / THREADS);
	dim3 blockSize(THREADS);


	Expression << < gridSize, blockSize, (3 * N) * sizeof(int) >> > (dA, x, dB, dC);
	Host_Expression(A, x, B, Ch);
	hipMemcpy(C, dC, sizeof(int) * N, hipMemcpyDeviceToHost);

	int razlika = CompareArrays(C, Ch);
	printf("Razlika %d\n", razlika);


	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}
