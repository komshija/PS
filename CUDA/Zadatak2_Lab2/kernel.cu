#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>

#define N 4
#define M 5
#define THREADS 32

__global__ void kernel(int* mat, int* max, int shsize)
{
    extern __shared__ int kolone[];
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int index = col + row * M; // row * total cols = prvi element u nekom redu, a + col pomeraj u tom redu 
    int localIndex = threadIdx.x + THREADS * threadIdx.y;
    
    if (row >= N || col >= M) return;

    //shared memoriju mapiramo na vise promenjivih
    int* colsh = kolone; // velicina shsize * shsize
    int* maxsh = &kolone[shsize*shsize]; // velicina shsize

    // ucitaju svi u svoju shared i inicijalizuje se max
    colsh[localIndex] = mat[index];
    maxsh[threadIdx.x] = 0;
    
    __syncthreads();

    // kroz atomicne operacije u shared nadje se max za svaki tile
    maxsh[threadIdx.y] = atomicMax(&maxsh[threadIdx.y], colsh[localIndex]);
    __syncthreads();


    max[blockIdx.y * (blockDim.x + gridDim.x)  + col] = maxsh[threadIdx.x];
}

void host(int *A, int* max) {

}

int main()
{
    int A[N][M], max[M], temp[N][M], maxh[M];
    int* Ad, *maxd;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            A[i][j] = i % 4 == 0 ? i * i - j : j * i;
        }
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf("%d ", A[i][j]);
        }
        printf("\n");
    }
    
    hipMalloc((void**)&Ad, sizeof(int) * N * M);
    hipMalloc((void**)&maxd, sizeof(int) * M * ((M + THREADS - 1) / THREADS));
    hipMemcpy(Ad, A, sizeof(int) * N * M, hipMemcpyHostToDevice);

    dim3 gridSize((N+THREADS-1)/THREADS,(M+THREADS-1)/THREADS);
    dim3 blockSize(THREADS,THREADS);
    int memSize = THREADS*(THREADS + 1);

    kernel << <gridSize, blockSize, memSize * sizeof(int) >> > (Ad, maxd, THREADS);
    
    hipMemcpy(temp, maxd, sizeof(int) * M * ((N + THREADS - 1) / THREADS), hipMemcpyDeviceToHost);

   // kernel << <1, blockSize, memSize * sizeof(int) >> > (maxd, maxd, THREADS);



    hipFree(Ad);
    hipFree(maxd);


    return 0;
}